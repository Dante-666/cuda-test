#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdio.h>

#include <FreeImage.h>

using namespace std;

struct RGB_24 {
    unsigned char r;
    unsigned char g;
    unsigned char b;
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void separateChannels(RGB_24* d_in, unsigned char* d_r, unsigned char* d_g, unsigned char* d_b, int numRows, int numCols) {
    
    if (blockIdx.x == (int) numCols/blockDim.x && threadIdx.x + blockIdx.x * blockDim.x >= numCols) return;
    else if (blockIdx.y == (int) numRows/blockDim.y && threadIdx.y + blockIdx.y * blockDim.y >= numRows) return;

    unsigned long toffset = threadIdx.x + threadIdx.y * numCols;
    unsigned long boffset = blockIdx.y * blockDim.x * numCols + blockDim.y * blockIdx.x;

    unsigned long id = toffset + boffset;

    d_r[id] = d_in[id].r;
    d_g[id] = d_in[id].g;
    d_b[id] = d_in[id].b;
}

__global__ void gaussBlur(RGB_24* d_out, unsigned char* d_r, unsigned char* d_g, unsigned char* d_b, int numRows, int numCols) {
    
    if (blockIdx.x == (int) numCols/blockDim.x && threadIdx.x + blockIdx.x * blockDim.x >= numCols) return;
    else if (blockIdx.y == (int) numRows/blockDim.y && threadIdx.y + blockIdx.y * blockDim.y >= numRows) return;
    
    unsigned long toffset = threadIdx.x + threadIdx.y * numCols;
    unsigned long boffset = blockIdx.y * blockDim.y * numCols + blockDim.x * blockIdx.x;

    unsigned long id = toffset + boffset;
    __shared__ RGB_24 pixels[34*34];
    unsigned long poffset = (blockDim.x + 2) * (threadIdx.y + 1) + threadIdx.x + 1;
    pixels[poffset].r = d_r[id];
    pixels[poffset].g = d_g[id];
    pixels[poffset].b = d_b[id];

    unsigned int t_poffset = poffset;
    unsigned long tid = id;
    
    if (id == 0) {
        t_poffset--;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset -= blockDim.x + 2;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset++;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
    } else if (id == numCols - 1) {
        t_poffset++;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset -= blockDim.x + 2;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset--;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
    } else if (id == numCols * (numRows - 1)) {
        t_poffset--;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset += blockDim.x + 2;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset++;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
    } else if (id == numCols * numRows - 1) {
        t_poffset++;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset += blockDim.x + 2;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        t_poffset--;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
    } else if (id < numCols) {
        t_poffset -= blockDim.x + 2;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        if (threadIdx.x == 0) {
            t_poffset--;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset += blockDim.x + 2;
            tid--;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        } else if (threadIdx.x == blockDim.x - 1) {
            t_poffset++;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset += blockDim.x + 2;
            tid++;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        }
    } else if (id % numCols == 0) {
        t_poffset--;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        if (threadIdx.y == blockDim.y - 1) {
            t_poffset += blockDim.x + 2;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset++;
            tid += numCols;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        } else if (threadIdx.y == 0) {
            t_poffset -= blockDim.x + 2;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset++;
            tid -= numCols;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        }

    } else if (id % numCols == numCols - 1) {
        t_poffset++;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        if (threadIdx.y == blockDim.y - 1) {
            t_poffset += blockDim.x + 2;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset--;
            tid -= numCols;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        } else if (threadIdx.y == 0) {
            t_poffset -= blockDim.x + 2;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset--;
            tid -= numCols;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        }
    } else if (id > numCols * (numRows - 1)) {
        t_poffset += blockDim.x + 2;
        pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
        if (threadIdx.x == 0) {
            t_poffset--;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset -= blockDim.x + 2;
            tid--;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        } else if (threadIdx.x == blockDim.x - 1) {
            t_poffset++;
            pixels[t_poffset].r = pixels[t_poffset].g = pixels[t_poffset].b = 0;
            t_poffset -= blockDim.x + 2;
            tid++;
            pixels[t_poffset].r = d_r[tid];
            pixels[t_poffset].g = d_g[tid];
            pixels[t_poffset].b = d_b[tid];
        }
    } else if (threadIdx.x == 0 && threadIdx.y == 0) {
        t_poffset--;
        tid--;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
        
        t_poffset -= blockDim.x + 2;
        tid -= numCols;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];

        t_poffset++;
        tid++;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    } else if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0) {
        t_poffset++;
        tid++;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
        
        t_poffset -= blockDim.x + 2;
        tid -= numCols;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];

        t_poffset--;
        tid--;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    } else if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1) {
        t_poffset--;
        tid--;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
        
        t_poffset += blockDim.x + 2;
        tid += numCols;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];

        t_poffset++;
        tid++;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    } else if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1) {
        t_poffset++;
        tid++;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
        
        t_poffset += (blockDim.x + 2);
        tid += numCols;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];

        t_poffset--;
        tid--;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    } else if (threadIdx.y == 0) {
        t_poffset -= blockDim.x + 2;
        tid -= numCols;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    } else if (threadIdx.x == 0) {
        t_poffset--;
        tid--;
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    } else if (threadIdx.y == blockDim.y - 1) {
        t_poffset += blockDim.x + 2;
        tid += numCols;    
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    } else if (threadIdx.x == blockDim.x - 1) {
        t_poffset++;
        tid++;    
        pixels[t_poffset].r = d_r[tid];
        pixels[t_poffset].g = d_g[tid];
        pixels[t_poffset].b = d_b[tid];
    }

    __syncthreads();

    float r, g, b;
    r = 0.147761f * float(pixels[poffset].r) +
        0.118318f * (float(pixels[poffset+1].r) 
                + float(pixels[poffset-1].r) 
                + float(pixels[poffset+blockDim.x+2].r) 
                + float(pixels[poffset-blockDim.x-2].r)) +
        0.0947416f * (float(pixels[poffset+blockDim.x+3].r) 
                + float(pixels[poffset+blockDim.x+1].r) 
                + float(pixels[poffset-blockDim.x-3].r) 
                + float(pixels[poffset-blockDim.x-1].r));
    g = 0.147761f * float(pixels[poffset].g) +
        0.118318f * (float(pixels[poffset+1].g) 
                + float(pixels[poffset-1].g) 
                + float(pixels[poffset+blockDim.x+2].g) 
                + float(pixels[poffset-blockDim.x-2].g)) +
        0.0947416f * (float(pixels[poffset+blockDim.x+3].g) 
                + float(pixels[poffset+blockDim.x+1].g) 
                + float(pixels[poffset-blockDim.x-3].g) 
                + float(pixels[poffset-blockDim.x-1].g));
    b = 0.147761f * float(pixels[poffset].b) +
        0.118318f * (float(pixels[poffset+1].b) 
                + float(pixels[poffset-1].b) 
                + float(pixels[poffset+blockDim.x+2].b) 
                + float(pixels[poffset-blockDim.x-2].b)) +
        0.0947416f * (float(pixels[poffset+blockDim.x+3].b) 
                + float(pixels[poffset+blockDim.x+1].b) 
                + float(pixels[poffset-blockDim.x-3].b) 
                + float(pixels[poffset-blockDim.x-1].b));
    
    /*if (id == 0) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id+1] + (float) d_r[id+numCols]) + 0.0947416f * (float) d_r[id+numCols+1];
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id+1] + (float) d_g[id+numCols]) + 0.0947416f * (float) d_g[id+numCols+1];
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id+1] + (float) d_b[id+numCols]) + 0.0947416f * (float) d_b[id+numCols+1];
    } else if (id == numCols - 1) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id-1] + (float) d_r[id+numCols]) + 0.0947416f * (float) d_r[id+numCols-1];
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id-1] + (float) d_g[id+numCols]) + 0.0947416f * (float) d_g[id+numCols-1];
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id-1] + (float) d_b[id+numCols]) + 0.0947416f * (float) d_b[id+numCols-1];
    } else if (id == numCols * (numRows - 1)) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id+1] + (float) d_r[id-numCols]) + 0.0947416f * (float) d_r[id-numCols+1];
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id+1] + (float) d_g[id-numCols]) + 0.0947416f * (float) d_g[id-numCols+1];
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id+1] + (float) d_b[id-numCols]) + 0.0947416f * (float) d_b[id-numCols+1];
    } else if (id == numCols * numRows - 1) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id-1] + (float) d_r[id-numCols]) + 0.0947416f * (float) d_r[id-numCols-1];
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id-1] + (float) d_g[id-numCols]) + 0.0947416f * (float) d_g[id-numCols-1];
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id-1] + (float) d_b[id-numCols]) + 0.0947416f * (float) d_b[id-numCols-1];
    } else if (id < numCols) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id-1] + (float) d_r[id+1] + 
            (float) d_r[id+numCols]) + 0.0947416f * ((float) d_r[id+numCols-1] + (float) d_r[id+numCols+1]);
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id-1] + (float) d_g[id+1] +
            (float) d_g[id+numCols]) + 0.0947416f * ((float) d_g[id+numCols-1] + (float) d_g[id+numCols+1]);
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id-1] + (float) d_b[id+1] + 
            (float) d_b[id+numCols]) + 0.0947416f * ((float) d_b[id+numCols-1] + (float) d_b[id+numCols+1]);
    } else if (id % numCols == 0) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id+1] + (float) d_r[id+numCols] +
            (float) d_r[id-numCols]) + 0.0947416f * ((float) d_r[id+numCols+1] + (float) d_r[id-numCols+1]);
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id+1] + (float) d_g[id+numCols] +
            (float) d_g[id-numCols]) + 0.0947416f * ((float) d_g[id+numCols+1] + (float) d_g[id-numCols+1]);
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id+1] + (float) d_b[id+numCols] +
            (float) d_b[id-numCols]) + 0.0947416f * ((float) d_b[id+numCols+1] + (float) d_b[id-numCols+1]);
    } else if (id % numCols == numCols - 1) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id-1] + (float) d_r[id+numCols] + 
            (float) d_r[id-numCols]) + 0.0947416f * ((float) d_r[id+numCols-1] + (float) d_r[id-numCols-1]);
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id-1] + (float) d_g[id+numCols] + 
            (float) d_g[id-numCols]) + 0.0947416f * ((float) d_g[id+numCols-1] + (float) d_g[id-numCols-1]);
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id-1] + (float) d_b[id+numCols] + 
            (float) d_b[id-numCols]) + 0.0947416f * ((float) d_b[id+numCols-1] + (float) d_b[id-numCols-1]);
    } else if (id > numCols * (numRows - 1)) {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id-1] + (float) d_r[id+1] +
            (float) d_r[id-numCols]) + 0.0947416f * ((float) d_r[id-numCols-1] + (float) d_r[id-numCols+1]);
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id-1] + (float) d_g[id+1] +
            (float) d_g[id-numCols]) + 0.0947416f * ((float) d_g[id-numCols-1] + (float) d_g[id-numCols+1]);
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id-1] + (float) d_b[id+1] + 
            (float) d_b[id-numCols]) + 0.0947416f * ((float) d_b[id-numCols-1] + (float) d_b[id-numCols+1]);
    } else {
        r = 0.147761f * (float) d_r[id] + 0.118318f * ((float) d_r[id-1] + (float) d_r[id+1] + (float) d_r[id-numCols] + (float) d_r[id+numCols]) +
            0.0947416f * ((float) d_r[id-numCols-1] + (float) d_r[id-numCols+1] + (float) d_r[id+numCols-1] + (float) d_r[id+numCols+1]);
        g = 0.147761f * (float) d_g[id] + 0.118318f * ((float) d_g[id-1] + (float) d_g[id+1] + (float) d_g[id-numCols] + (float) d_g[id+numCols]) +
            0.0947416f * ((float) d_g[id-numCols-1] + (float) d_g[id-numCols+1] + (float) d_g[id+numCols-1] + (float) d_g[id+numCols+1]);
        b = 0.147761f * (float) d_b[id] + 0.118318f * ((float) d_b[id-1] + (float) d_b[id+1] + (float) d_b[id-numCols] + (float) d_b[id+numCols]) +
            0.0947416f * ((float) d_b[id-numCols-1] + (float) d_b[id-numCols+1] + (float) d_b[id+numCols-1] + (float) d_b[id+numCols+1]);
    }*/

    d_out[id].r = r ; d_out[id].g = g ; d_out[id].b = b;
    /*d_out[id].r = pixels[poffset].r; 
    d_out[id].g = pixels[poffset].g;
    d_out[id].b = pixels[poffset].b;*/
}

int main(int argc, char** argv) {
    
    if (argc < 2 || argc > 2) return -1;
    FreeImage_Initialise(); 
    FREE_IMAGE_FORMAT format = FreeImage_GetFileType(argv[1]);
    FIBITMAP* immap = FreeImage_Load(format, argv[1]);

    int numRows = FreeImage_GetHeight(immap);
    int numCols = FreeImage_GetWidth(immap);
    int pitch = FreeImage_GetPitch(immap);

    
    RGB_24* h_in = new RGB_24[numRows * numCols];
    RGB_24* h_out = new RGB_24[numRows * numCols];
    RGB_24* d_in;
    unsigned char* d_r;
    unsigned char* d_g;
    unsigned char* d_b;
    RGB_24* d_out;
    
    gpuErrchk(hipMalloc((void **) &d_in, sizeof(RGB_24) * numRows * numCols));

    gpuErrchk(hipMalloc((void **) &d_r, sizeof(unsigned char) * numRows * numCols));
    gpuErrchk(hipMalloc((void **) &d_g, sizeof(unsigned char) * numRows * numCols));
    gpuErrchk(hipMalloc((void **) &d_b, sizeof(unsigned char) * numRows * numCols));
    
    FREE_IMAGE_TYPE type = FreeImage_GetImageType(immap);
    int i = 0;
    if(type == FIT_BITMAP) {
        BYTE* bits = (BYTE*)FreeImage_GetBits(immap);
        for(int y = 0; y < numRows; y++) {
            BYTE* pixel = (BYTE *) bits;
            for(int x = 0; x < numCols; x++) {
                h_in[i].r = pixel[FI_RGBA_RED];
                h_in[i].g = pixel[FI_RGBA_GREEN];
                h_in[i++].b = pixel[FI_RGBA_BLUE];
                pixel += 3;
            }
            bits += pitch;
        }
    } 
    
    gpuErrchk(hipMemcpy(d_in, h_in, numRows * numCols * sizeof(RGB_24), hipMemcpyHostToDevice));
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    
    cout<<sizeof(float)<<endl;
    separateChannels<<<dim3(ceil(numCols/32.0), ceil(numRows/32.0), 1), dim3(32, 32, 1)>>>(d_in, d_r, d_g, d_b, numRows, numCols);
    
    gpuErrchk(hipMalloc((void **) &d_out, sizeof(RGB_24) * numRows * numCols));
   
    gpuErrchk(hipFree(d_in));
    
    gaussBlur<<<dim3(ceil(numCols/32.0), ceil(numRows/32.0), 1), dim3(32, 32, 1)>>>(d_out, d_r, d_g, d_b, numRows, numCols);
    gpuErrchk(hipPeekAtLastError());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    cout<<"Time taken "<<time<<endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    gpuErrchk(hipMemcpy(h_out, d_out, numRows * numCols * sizeof(RGB_24), hipMemcpyDeviceToHost));
   
    gpuErrchk(hipFree(d_out));
    gpuErrchk(hipFree(d_r));
    gpuErrchk(hipFree(d_g));
    gpuErrchk(hipFree(d_b));
    
    //hipDeviceSynchronize();

    BYTE* bits = (BYTE*)FreeImage_GetBits(immap);
    i = 0;
    if(type == FIT_BITMAP) {
        for(int y = 0; y < numRows; y++) {
            BYTE* pixel = (BYTE *) bits;
            for(int x = 0; x < numCols; x++) {
                pixel[FI_RGBA_RED] = h_out[i].r;
                pixel[FI_RGBA_GREEN] = h_out[i].g;
                pixel[FI_RGBA_BLUE] = h_out[i++].b;
                pixel += 3;
            }
            bits += pitch;
        }
    }

    FreeImage_Save(FIF_PNG, immap, "blur.png", JPEG_DEFAULT);
    FreeImage_DeInitialise();

    return 0; 
}
