//#include <cuda.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void  square(const float *d_in, float *d_out){
    d_out[threadIdx.x] = d_in[threadIdx.x]*d_in[threadIdx.x];
}

int main() {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    float *h_in, *h_out, *d_in, *d_out;
   
    h_in = (float *) malloc(ARRAY_BYTES);
    h_out = (float *) malloc(ARRAY_BYTES);

    for (int i = 0; i < ARRAY_SIZE;) {
        h_in[i] = (float) ++i;
        h_out[i-1] = h_in[i-1];
    }

    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    square<<<1, ARRAY_SIZE>>>(d_in, d_out);
    
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
   
    for (int i=0; i<ARRAY_SIZE; i++){
        printf("%f\t", h_out[i]);
    }
    hipFree(d_in);
    hipFree(d_out);
}
